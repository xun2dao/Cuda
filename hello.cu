#include<hip/hip_runtime.h>
#include<cstdio>
__device__ __host__ void say_hello(){ // 通过__device__ 和__host__把say_hello同时定义在GPU和CPU上
#ifdef __CUDA_ARCH__   // 我们可以通过__CUDA_ARCH__这个宏区分一个被声明为__device__ __host__的函数此时究竟在哪里运行。
  printf("Say Hello GPU : %d\n", __CUDA_ARCH__);
#else
  printf("Say Hello CPU!\n");
#endif
}


