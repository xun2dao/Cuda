#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<cstdio>
#include<cmath>
#include<vector>
#include<type_traits>
#include<hip/hip_runtime_api.h>
#include"cudaAllocator.h"
#include"Tick.h"
//#include<thrust/universal_vector.h>

__device__ __host__ void say_hello();

template<typename Func>
__global__ void kernel(int n, Func func){
  for(int i = blockIdx.x * blockDim.x + threadIdx.x;  i < n; i += blockDim.x*gridDim.x){
    func(i);
  }
}

__global__ void Iterate2D(float* devPtr, size_t pitch, int width, int height){
  printf("Pitch is %d\n", pitch);
  printf("width is %d\n", width);
  printf("height is %d\n", height);
  for(int i = 0; i < height; ++i){
    float* row = (float*)((char*)devPtr + i * pitch);
    for(int j = 0; j < width; ++j){
      float element = row[j];
    }
  }
}

__global__ void Iterate3D(hipPitchedPtr ptr, int width, int height, int depth){
  char* devptr = (char*)ptr.ptr;
  size_t pitch = ptr.pitch;
  size_t slicePitch = pitch * height;
  printf("Pitch is %d\n", pitch);
  printf("slicePitch is %d\n", slicePitch);
  printf("width is %d\n", width);
  printf("heigth is %d\n", height);
  printf("depth is %d\n", depth);


  for(int i = 0; i < depth; ++i){
    char* slice =  devptr + i *slicePitch;
    for(int j = 0; j < height; ++j){
      float* row = (float*)(slice + j * pitch);
      for(int z = 0; z < width; ++z){
        float element = row[z];
      }
    }
  }
}

void device3DMemory(){
  int width = 64, height = 64, depth = 64;
  hipExtent extent = make_hipExtent(width, height, depth);
  hipPitchedPtr pitchedPtr;
  hipMalloc3D(&pitchedPtr,extent);
  Iterate3D<<<100, 100>>>(pitchedPtr, width, height, depth);
  hipDeviceSynchronize();
  
}


 __device__ float* devsrc;
int main(){
  //int *pret;
  // 在显存上分配数据
  /*
  hipError_t merr = hipMalloc(&pret, sizeof(int));
  kernel<<<1, 5>>>(pret);
  hipError_t err = hipDeviceSynchronize();
  int ret;
  // 从显存拷贝的内存
  hipMemcpy(&ret, pret, sizeof(int), hipMemcpyDeviceToHost);

  hipFree(pret);
  
  printf("Cuda Get Error: %d\n", err);
  printf("Cuda Error is %s\n", hipGetErrorName(err));
  printf("ret's value is %d\n",ret);
  */
  /*
  hipMallocManaged(&pret, sizeof(int)); // 统一内存地址分配
  kernel<<<1,1>>>(pret);
  hipDeviceSynchronize(); // 使用统一地址分配，不要忘记同步。
  printf("Ret value is %d\n", *pret);
  hipFree(pret);
  */


  // 对数组进行赋值
 // int n = 1 << 25;
 // int threads = 256;
 // int blocks = n / threads;
 // std::vector<float, cudaAllocator<float>> arr(n);
 // std::vector<float> cpu(n);
 // {
 //   Timer timer("GPU");
 //   kernel<<<20, 256>>>(n, [arr = arr.data()]__device__(int i){
 //     arr[i] = __sinf(i);
 //     });
 //   hipDeviceSynchronize(); 
 //   hipDeviceReset(); // 摧毁当前的CUDA上下文
 // }

 // {
 //   Timer timer("CPU");
 //   for(int i = 0; i < n; ++i) {
 //     cpu[i] = sinf(i);
 //   }
 // }
  /*
  int width = 64, height = 64;
  size_t pitch;
  float* devPtr;
  hipMallocPitch(&devPtr, &pitch, width*sizeof(float), height);
  Iterate2D<<<100, 128>>>(devPtr, pitch, width, height);
  hipDeviceSynchronize();
  hipFree(devPtr);
  */
  
  //device3DMemory();
  
  float* src;
  hipMalloc(&src, sizeof(float)*256);
  hipMemcpyToSymbol(HIP_SYMBOL(devsrc), &src, sizeof(src));  // 针对全局的device变量的数据
  return 0;
}
