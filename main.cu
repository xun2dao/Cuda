#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<cstdio>
#include<cmath>
#include<vector>
#include<type_traits>
#include"cudaAllocator.h"


__device__ __host__ void say_hello();

template<typename Func>
__global__ void kernel(int n, Func func){
  for(int i = blockIdx.x * blockDim.x + threadIdx.x;  i < n; i += blockDim.x*gridDim.x){
    func(i);
  }
}


int main(){
  //int *pret;
  // 在显存上分配数据
  /*
  hipError_t merr = hipMalloc(&pret, sizeof(int));
  kernel<<<1, 5>>>(pret);
  hipError_t err = hipDeviceSynchronize();
  int ret;
  // 从显存拷贝的内存
  hipMemcpy(&ret, pret, sizeof(int), hipMemcpyDeviceToHost);

  hipFree(pret);
  
  printf("Cuda Get Error: %d\n", err);
  printf("Cuda Error is %s\n", hipGetErrorName(err));
  printf("ret's value is %d\n",ret);
  */
  /*
  hipMallocManaged(&pret, sizeof(int)); // 统一内存地址分配
  kernel<<<1,1>>>(pret);
  hipDeviceSynchronize(); // 使用统一地址分配，不要忘记同步。
  printf("Ret value is %d\n", *pret);
  hipFree(pret);
  */


  // 对数组进行赋值
  int n = 100;
  int threads = 256;
  int blocks = n / threads;
  std::vector<int, cudaAllocator<int>> arr(n);
  kernel<<<1, 10>>>(n, [arr = arr.data()]__device__(int i){
    arr[i] = i;
    printf("arr[%d] = %d\n", i, arr[i]);
    });
  hipDeviceSynchronize(); 
  return 0;
}
